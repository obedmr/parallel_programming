#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 40000

// Multiplicacion de matrices

__global__ void multiply_matrix(float *c, float *a, float *b, int rows, int cols){

  int N = rows * cols; 
  int col = blockIdx.x*blockDim.x + threadIdx.x;  
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = row * N * col;
  c[idx] = a[idx] * b[idx];

}

//función principal
int main(void){
  float *A_h, *B_h, *C_h;
  float *A_d, *B_d, *C_d;

  int n_rows = 100;
  int n_cols = 100;
  int N = n_rows * n_cols;

  // Asignando memoria en Host
  A_h = (float *) * malloc(N * sizeof(float));
  B_h = (float *) * malloc(N * sizeof(float));
  C_h = (float *) * malloc(N * sizeof(float));

  //asignar memoria en la GPU
  hipMalloc((void**)&A_d,N*sizeof(float));
  hipMalloc((void**)&B_d,N*sizeof(float));
  hipMalloc((void**)&C_d,N*sizeof(float));

  //Llenar datos a los arreglos 'a' y 'b' en la CPU
  for(int i=0; i<N;i++){
    A_h[i]=i;
    B_h[i]=i+1;
  }

  //copiar el arreglo 'a' y 'b' en la GPU
  hipMemcpy(A_d,A_h,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(B_d,B_h,N*sizeof(float),hipMemcpyHostToDevice);


  dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
  dim3 n_blocks((ceil(float(block_size.x/n_cols)), ceil(float(block_size.y/n_rows)));

  //lanzar elkernel
  multiply_matrix<<<n_blocks, block_size>>>(C_d, A_d, B_d, n_rows, n_cols);

  //copiar el arreglo 'c' de la GPU a la CPU
  hipMemcpy(C_h,C_d,N*sizeof(float),hipMemcpyDeviceToHost);

  //Desplegar el resultado
  for(int i=0; i<N; i++){
    printf("%.2f*%.2f=%.2f\n",A_h[i],B_h[i],C_h[i]);
  }

  free(A_h);
  free(B_h);
  free(C_h);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  return 0;
  
} 
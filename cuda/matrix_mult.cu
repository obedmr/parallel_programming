
#include <hip/hip_runtime.h>
#include<stdio.h>

#define BLOCK_SIZE 3

// Multiplicacion de matrices

__global__ void multiply_matrix(float *c, float *a, float *b, int rows, int cols){

  int col = blockIdx.x*blockDim.x + threadIdx.x;  
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = row * cols + col;

    float sum = 0;
    int i;
    for(i=0; i<cols; i++)
      sum+=a[row * cols + i] * b[cols * i + col];
      
    c[idx] = sum;

}

//función principal
int main(void){
  float *A_h, *B_h, *C_h;
  float *A_d, *B_d, *C_d;

  int n_rows = 3;
  int n_cols = 3;
  int N = n_rows * n_cols;

  // Asignando memoria en Host
  A_h = (float *) malloc(N * sizeof(float));
  B_h = (float *) malloc(N * sizeof(float));
  C_h = (float *) malloc(N * sizeof(float));

  //asignar memoria en la GPU
  hipMalloc((void**)&A_d,N*sizeof(float));
  hipMalloc((void**)&B_d,N*sizeof(float));
  hipMalloc((void**)&C_d,N*sizeof(float));

  //Llenar datos a los arreglos 'a' y 'b' en la CPU
  for(int i=0; i<N;i++){
    A_h[i]=i;
    B_h[i]=i+1;
  }

  //copiar el arreglo 'a' y 'b' en la GPU
  hipMemcpy(A_d,A_h,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(B_d,B_h,N*sizeof(float),hipMemcpyHostToDevice);


  dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
  dim3 n_blocks(ceil(block_size.x/n_cols), ceil(block_size.y/n_rows));

  printf("dim3 = n_blocks(%f, %f)\n", ceil(block_size.x/n_cols), ceil(block_size.y/n_rows) ); 

  //lanzar elkernel
  multiply_matrix<<<1, block_size>>>(C_d, A_d, B_d, n_rows, n_cols);

  //copiar el arreglo 'c' de la GPU a la CPU
  hipMemcpy(C_h,C_d,N*sizeof(float),hipMemcpyDeviceToHost);

  //Desplegar el resultado
  int i;
  for(i=0; i<N; i++){
    printf("%.2f\n",C_h[i] );
  }

  free(A_h);
  free(B_h);
  free(C_h);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  return 0;  
} 

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void holaCUDA(float e) {
  printf("Hola, soy el hilo %d del bloque %d con valor pi->%f\n", threadIdx.x,blockIdx.x,e);
}

int main(int argc, char **argv){
  holaCUDA<<<8,4>>>(3.1416);
  hipDeviceReset(); //Esta llamada reinicializa el device
  return 0;
}